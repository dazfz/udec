#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <hip/hip_cooperative_groups.h>
#include <algorithm>
#include <random>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cooperative_groups;


// Kernel para encontrar el maximo
__device__ int reduce_max(thread_group g, int *temp, int val) {
	int tid = g.thread_rank();

	// toma una mitad para compararla con la otra, luego la mitad de la primera mitad, y asi
	for (int i = blockDim.x/2; i>0; i /= 2){
		// cada hilo toma un valor, se espera a que todos terminen
		temp[tid] = val;	
		g.sync();

		// cada hilo compara su valor con el de la otra mitad respectivo y guarda el maximo, se espera a que todos terminen
		if(tid<i) val = max(val, temp[tid + i]);
		g.sync();
	}

	return val;
}


// Kernel inicio, cooperative groups
__global__ void reduce(int *maximos, int *input){
	// memoria compartida del stream
    extern __shared__ int temp[];

	// id del thread
    int id = blockIdx.x*blockDim.x + threadIdx.x;		

	// grupo cooperativo
    thread_group g = this_thread_block();

	// llamada al reduce 
    int block_max = reduce_max(g, temp, input[id]);

	// se guarda el valor maximo, la primera hebra es la encargada en este caso
    if (threadIdx.x == 0) atomicMax(maximos, block_max);
}


int main(int argc, char *argv[]) {
	// arreglos, tamaño
	int n = 3, k1 = 5, p = 0, bloques = -1;
	for(int i=0; i<argc; i++){
		if( !strcmp(argv[i], "-n" ) ) n = atoi(argv[i+1]);
		if( !strcmp(argv[i], "-k" ) ) k1 = atoi(argv[i+1]);
		if( !strcmp(argv[i], "-p" ) ) p = 1;
		if( !strcmp(argv[i], "-b" ) ) bloques = atoi(argv[i+1]);
	}


	int k = pow(2, ceil(log(k1)/log(2)));

	// memoria
	vector<int*> arreglosDst(n), arreglosSrc(n), maximos(n);
    for (int i = 0; i < n; i++) {
        hipHostMalloc(&arreglosSrc[i], k * sizeof(int));
        hipMalloc(&arreglosDst[i], k * sizeof(int));
        hipHostMalloc(&maximos[i], sizeof(int));
    }
	

	// creacion arreglos y relleno de ceros para que sea potencia de 2
	for(int i=0; i<n; i++){
		for(int j=0; j<k; j++){
			if(j < k1) arreglosSrc[i][j] = j;
			else arreglosSrc[i][j] = 0;
		}
		shuffle(arreglosSrc[i], arreglosSrc[i] + k1, mt19937{random_device{}()});
	}

	// print arreglos
	if(p){
		for(int i=0; i<n; i++){
			for(int j=0; j<k; j++){
				cout << arreglosSrc[i][j] << " ";
			}
			cout << endl;
		}
	}

	// streams
	vector<hipStream_t> stream(n);
	for (int i=0; i<n; i++) hipStreamCreate(&stream[i]);

	// calculo para no superar el limite de hebras del bloque
	if(bloques == -1) bloques = 2 + k/1024;
	int hebras = (k + bloques - 1) / bloques;
	int sharedBytes = hebras * sizeof(int);

	cout << "Bloques:    " << bloques << endl;
	cout << "Hebras:     " << hebras << endl;
	cout << "sharedbits: " << sharedBytes << endl;
	
	for (int i=0; i<n; i++) {
		// copia arreglo host a gpu
		hipMemcpyAsync(arreglosDst[i], arreglosSrc[i], k * sizeof(int), hipMemcpyHostToDevice, stream[i]);
		
		// kernel
		reduce<<<bloques, hebras, sharedBytes, stream[i]>>>(maximos[i], arreglosDst[i]);
	}
	
	// Sincronizacion de los streams
	hipDeviceSynchronize();	

	// print de los maximos
    for (int i = 0; i < n; i++) cout << maximos[i][0] << " ";

	// liberacion memoria y strems
	for(int i=0; i<n; i++){
		hipFree(arreglosDst[i]);
		hipHostFree(arreglosSrc[i]);
		hipStreamDestroy(stream[i]);
	}
	
	return 0;
}

